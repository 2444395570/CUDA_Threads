#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <memory>

#define N 50000           //�������ж���Ԫ�ص�����

__global__ void gpuAdd(int* d_a, int* d_b, int* d_c) {
	//������ǰ�ں˵�����
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid<N)
	{
		d_c[tid] = d_a[tid] + d_b[tid];
		tid += blockDim.x * gridDim.x;
	}
}


int main(void) {
	//�����������豸������
	int h_a[N], h_b[N], h_c[N];
	int* d_a, * d_b, * d_c;

	//���豸�Ϸ����ڴ�
	hipMalloc((void**)&d_a, N * sizeof(int));
	hipMalloc((void**)&d_b, N * sizeof(int));
	hipMalloc((void**)&d_c, N * sizeof(int));

	//��ʼ����������
	for (int i = 0; i < N; i++)
	{
		h_a[i] = 2 * i * i;
		h_b[i] = i;
	}

	hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
	
	//�ں˵���
	gpuAdd << <512, 512 >> > (d_a, d_b, d_c);
	hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
	//�¾����ȷ���ں�ִ���ڼ������֮ǰ

	hipDeviceSynchronize();
	int Correct = 1;
	printf("Vector addition an GPU\n");
	for (int i = 0; i < N; i++)
	{
		if ((h_a[i] + h_b[i] != h_c[i])) {
			Correct = 0;
		}
	}
	if (Correct == 1)
	{
		printf("GPU has computed Sum Correctly\n");
	}
	else
	{
		printf("There is an Error in GPU Compuation\n");
	}

	//�ͷ��ڴ�
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}